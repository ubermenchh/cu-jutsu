// Vector Addition in CUDA using Pinned Memory
//
// Blog: https://giahuy04.medium.com/pinned-memory-5d408b72241d
        

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

__global__ void vector_add(int* a, int* b, int* c, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Thread ID
    
    if (tid < N) // Boundary check
        c[tid] = a[tid] + b[tid];
}

void verify_result(int *a, int *b, int *c, int N) {
  for (int i = 0; i < N; i++) {
    assert(c[i] == a[i] + b[i]);
  }
}

int main() {
    const int N = 1 << 26;
    size_t bytes = sizeof(int) * N;
    
    int* h_a, *h_b, *h_c;
    
    // allocate pinned memory
    hipHostMalloc(&h_a, bytes, hipHostMallocDefault);
    hipHostMalloc(&h_b, bytes, hipHostMallocDefault);
    hipHostMalloc(&h_c, bytes, hipHostMallocDefault);
    
    // init vectors
    for (int i = 0; i < N; i++) {
        h_a[i] = rand() % 100;
        h_b[i] = rand() % 100;
    }
    
    // allocate memory on the device
    int* d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
    
    // copy data from host to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, bytes, hipMemcpyHostToDevice);
    
    int THREADS = 1 << 10; // 1024 threads
    int BLOCKS = (N + THREADS - 1) / THREADS;
    
    vector_add <<< BLOCKS, THREADS >>> (d_a, d_b, d_c, N);
    
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
    
    verify_result(h_a, h_b, h_c, N);
    
    // free pinned memory
    hipHostFree(h_a); hipHostFree(h_b); hipHostFree(h_c);
    
    // free device memory
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    
    printf("SUCCESS!!!");
    return 0;
}
